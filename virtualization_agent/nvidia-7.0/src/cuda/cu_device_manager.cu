#include "hip/hip_runtime.h"
#include <cu_device_manager.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>

__global__ void found_gpu()
{
  printf("Found GPU! \n"); 
  //__syncthreads();
}

void check_error()
{
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    printf("Error: %s\n", hipGetErrorString(err));
}

void h_found_gpu()
{
  found_gpu<<<1,1>>>();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
__global__ void cu_pw_method(int method, T * in, T * out, T bias, size_t sz)
{
  // Get our global thread ID
  int id = blockIdx.x*blockDim.x+threadIdx.x;
  // Make sure we do not go out of bounds
  if( method == 1) if (id < sz) out[id] = in[id] + bias;
  if( method == 2) if (id < sz) out[id] = in[id] * bias;
  if( method == 3) if (id < sz) out[id] = in[id] / bias;
}

template<typename T>
void h_pw_method( int method, const T * in, T * out, T bias, size_t len)
{
  T * _in, *_out;
  int blockSize, gridSize;
  size_t bytes = len*sizeof(T);
  blockSize = 32;
  // Allocate memory for each vector on GPU
  hipMalloc(&_in, bytes );
  hipMalloc(&_out, bytes );
  // Copy host vectors to device
  hipMemcpy( _in, in, bytes, hipMemcpyHostToDevice);
  // Number of threads in each thread block
  // Number of thread blocks in grid
  gridSize = (int)ceil(len/blockSize);
  cu_pw_method<<<gridSize, blockSize>>>(method, _in, _out, bias, len );
  // Copy array back to host
  hipMemcpy( out, _out, bytes, hipMemcpyDeviceToHost );
  hipDeviceSynchronize();
  // Release device memory
  hipFree(_in);
  hipFree(_out);
}


template void h_pw_method( int method, const float * in, float * out, float bias, size_t len);
template void h_pw_method( int method, const double * in, double * out, double bias, size_t len);
template void h_pw_method( int method, const unsigned char * in, unsigned char * out, unsigned char bias, size_t len);
template void h_pw_method( int method, const char * in, char * out, char bias, size_t len);
template void h_pw_method( int method, const int * in, int * out, int bias, size_t len);
template void h_pw_method( int method, const unsigned int * in, unsigned int * out, unsigned int bias, size_t len);
template void h_pw_method( int method, const long * in, long * out, long bias, size_t len);
template void h_pw_method( int method, const unsigned long * in, unsigned long * out, unsigned long bias, size_t len);;
template void h_pw_method( int method, const unsigned long long * in, unsigned long long * out, unsigned long long bias, size_t len);
template void h_pw_method( int method, const long long * in, long long * out, long long bias, size_t len);
